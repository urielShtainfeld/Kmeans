#include "hip/hip_runtime.h"
#include"Header.h"


__global__ void setByTimeKernel(Point *arrOfPoints, float dt, int size)
{
	unsigned long id;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size)
	{
		arrOfPoints[id].x = arrOfPoints[id].x + (dt * arrOfPoints[id].Vx);
		arrOfPoints[id].y = arrOfPoints[id].y + (dt * arrOfPoints[id].Vy);
	}
}


hipError_t movePointInTime(Point *arrOfPoints, float dt, int size)
{
	Point *pointsArray = 0;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int numOfThreadsPerBlock;

	hipGetDeviceProperties(&prop, 0);
	numOfThreadsPerBlock = prop.maxThreadsPerBlock;
	int numOfBlock = (size / numOfThreadsPerBlock);
	if (size % numOfThreadsPerBlock != 0)
	{
		numOfBlock += 1;
	}

	//check for the cuda device if have error reaching him
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}
	//check if there are error in malloc space
	cudaStatus = hipMalloc((void**)&pointsArray, size * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}
	// check if copy there is error in copy from host
	cudaStatus = hipMemcpy(pointsArray, arrOfPoints, size * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	setByTimeKernel << < numOfBlock, size >> >(pointsArray, dt, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(arrOfPoints, pointsArray, size * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}


}
void freePointsArrayByCuda(Point *arrOfPoints)
{
	hipFree(arrOfPoints);
}
