#include "hip/hip_runtime.h"
#include"Header.h"


__global__ void setByTimeKernel(Point *arrOfPoints, float dt, int size)
{
	int i, j;
	unsigned long id;
	i = threadIdx.x;
	j = blockIdx.x;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size)
	{
		arrOfPoints[id].x = arrOfPoints[id].x + (dt * arrOfPoints[id].Vx);
		arrOfPoints[id].y = arrOfPoints[id].y + (dt * arrOfPoints[id].Vy);
	}
}


hipError_t movePointInTime(Point *arrOfPoints, float dt, int size)
{
	Point *pointsArray = 0;
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int numOfThreadsPerBlock;

	hipGetDeviceProperties(&prop, 0);
	numOfThreadsPerBlock = prop.maxThreadsPerBlock;
	int numOfBlock = (size / numOfThreadsPerBlock);
	if (size % numOfThreadsPerBlock != 0)
	{
		numOfBlock += 1;
	}

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&pointsArray, size * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(pointsArray, arrOfPoints, size * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	//setKernel << <1, size >> >(p_Arry, from, to,size,dT);
	setByTimeKernel << < numOfBlock, size >> >(pointsArray, dt, size);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(arrOfPoints, pointsArray, size * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freePointsArrayByCuda(pointsArray);
		return cudaStatus;
	}


}
void freePointsArrayByCuda(Point *arrOfPoints)
{
	hipFree(arrOfPoints);
}
